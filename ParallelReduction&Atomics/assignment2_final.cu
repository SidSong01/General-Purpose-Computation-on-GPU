#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define DIM 512  // threads per block
#define N (1 << 24) // vector length 2^24

float *get_vec(unsigned int n) {
  float *vector = (float *)malloc(sizeof(float) * n);
  srand((unsigned int)time(NULL));
  float a = rand() % 100 + 1;
  for (int i = 0; i < n; i++) {
  vector[i] =(((float) rand() /(float)(RAND_MAX)) * a);     
  }
  return vector;
}

__global__ void kernel1(float* output, float *vector_1, float *vector_2, unsigned int n){
  __shared__ float smem[DIM];

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= n) return;


  int tid = threadIdx.x;
  smem[tid] = vector_1[idx] * vector_2[idx];
  __syncthreads();


  for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
    if(tid < stride) smem[tid] += smem[tid + stride];
    __syncthreads();
  }

  if(tid == 0) output[blockIdx.x] = smem[0];
}


__global__ void kernel2(float* result, float *vector_1, float *vector_2, int n) {
 
  __shared__ float smem[DIM];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx >= n) return;

  int tid = threadIdx.x;
  smem[tid] = vector_1[idx] * vector_2[idx];
  __syncthreads();

  for(int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if(tid < stride) smem[tid] += smem[tid + stride];
    __syncthreads();
  }
  if(tid == 0) atomicAdd(result, smem[0]);
}

float gpu_dot_product_1(float *vector_1, float *vector_2, int n) {
  hipEvent_t start, stop;
  float elapsed;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  unsigned int n_blocks = (n + DIM - 1) / DIM;
  float *d_output, *d_vector_1, *d_vector_2, *output;
  unsigned int output_size = n_blocks * sizeof(float),
               input_size = n * sizeof(float);
  float result = 0;

  hipMalloc(&d_output, output_size);
  hipMalloc(&d_vector_1, input_size);
  hipMalloc(&d_vector_2, input_size);

  hipMemcpy(d_vector_1, vector_1, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_vector_2, vector_2, input_size, hipMemcpyHostToDevice);

  
  hipEventRecord(start);
  kernel1<<<n_blocks, DIM>>>(d_output, d_vector_1, d_vector_2, n);
  hipEventRecord(stop); 
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  printf("     Kernel1 execution time: %fms\n", elapsed);
  

  output = (float*)malloc(output_size);
  hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);

  hipFree(d_output);
  hipFree(d_vector_1);
  hipFree(d_vector_2);

  for(unsigned int i = 0; i < n_blocks; i++) {
    result += output[i];
  }

  free(output);
  return result;

}


float gpu_dot_product_2(float *vector_1, float *vector_2, int n) {
  hipEvent_t start, stop;
  float elapsed;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  unsigned int n_blocks = (n + DIM - 1) / DIM;
  float *d_result, *d_vector_1, *d_vector_2, *result;
  unsigned int output_size = sizeof(float),
               input_size = n * sizeof(float);

  hipMalloc(&d_result, output_size);
  hipMalloc(&d_vector_1, input_size);
  hipMalloc(&d_vector_2, input_size);

  hipMemcpy(d_vector_1, vector_1, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_vector_2, vector_2, input_size, hipMemcpyHostToDevice);


  hipEventRecord(start);
  kernel2<<<n_blocks, DIM>>>(d_result, d_vector_1, d_vector_2, n);
  hipEventRecord(stop); 
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  printf("     Kernel2 execution time: %fms\n", elapsed);
  

  result = (float*)malloc(output_size);
  hipMemcpy(result, d_result, output_size, hipMemcpyDeviceToHost);

  hipFree(d_result);
  hipFree(d_vector_1);
  hipFree(d_vector_2);

  return *result;
}

int main(){
  float *a = get_vec(N);
  float *b = get_vec(N);

  hipEvent_t start, stop;
  float elapsed;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  printf("===================Kernel1=================\n");
  hipEventRecord(start);
  float r1 = gpu_dot_product_1(a, b, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  printf("     Result : %f\n", r1);
  printf("     Total time: %fms\n\n", elapsed);

  printf("===================Kernel2=================\n");
  hipEventRecord(start);
  float r2 = gpu_dot_product_2(a, b, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  printf("     Result : %f\n", r2);
  printf("     Total time: %fms\n\n", elapsed);

  free(a);
  free(b);
}
